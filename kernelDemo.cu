#include "hip/hip_runtime.h"

// Via My Test Solution Case03

#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include <math.h>		//  Check error 
#include <cmath>


// CPU  original ArrayAdd
void MyAdd(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++)
	{
		y[i] = x[i] + y[i];
	}
}

// malloc memory aimed at CPU
void mallocCPU(int n, float *x, float *y)
{
	x = new float[n];
	y = new float[n];
}

// free memory aimed at CPU function&vars
void freeCPU(float *x, float *y)
{
	delete[] x;
	delete[] y;
}

// GPU  My test
__global__ void MyAddGPU(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		y[i] = x[i] + y[i];
	}

	if (1 == index)
	{
		printf("blockDim.x = %d\n", blockDim.x);
		printf("gridDim.x = %d\n", gridDim.x);	// <<<gridDim.x, blockDim.x>>>
		
		//std::cout << "gridDim.x = " << gridDim.x << std::endl;
	}
	
}

// malloc memory aimed at GPU 
void mallocGPU(int n, float *x, float *y)
{
	hipMallocManaged(&x, n * sizeof(float));	
	hipMallocManaged(&y, n * sizeof(float));
}

// free memory aimed at GPU funcs&vars
void freeGPU(float *x, float *y)
{
	hipFree(x);
	hipFree(y);
}


// GPU demo  default
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


int main()
{
	
	// -------------------------------------------
	// My Test code part 
	// -------------------------------------------
	const unsigned int N = 1 << 20;	// almost 1M elements, 2^20 = 1048576
	float *x, *y;

	//mallocCPU(N, x, y);
	//mallocGPU(N, x, y);
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the CPU with for loop
	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// -------------------------------------------

	// Add vectors in parallel.

	// ---------------------------------------------
	// My test code part II 2 
	// ---------------------------------------------

	// run MyAdd on CPU 
	//MyAdd(N, x, y);

	// run MyAddGPU on the GPU
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	MyAddGPU <<<numBlocks, blockSize>>>(N, x, y);	// 
	//This type of loop in a CUDA kernel is often called a grid - stride loop
	
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// check for errors (as u know, all the values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
	{
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	}
	std::cout << "\nChecking errors:\n\n" << "maxError = " << maxError << std::endl;
	//std::cout << "Running datails:\n" << std::endl;
	//std::cout << "GirdDim.x = " << gridDim.x << std::endl;
	//std::cout << "blockDim.x = " << blockDim.x << std::endl;


	// ---------------------------------------------

	std::cout << "------------------------------------------" << std::endl;
	std::cout << "Device Properties, as follows:" << std::endl;

	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;
	
//	struct __device_builtin__ hipDeviceProp_t
//	{
//    char   name[256];                  /**< ASCII string identifying device */
//	size_t totalGlobalMem;             /**< Global memory available on device in bytes */
//	size_t sharedMemPerBlock;          /**< Shared memory available per block in bytes */
//	int    regsPerBlock;               /**< 32-bit registers available per block */
//	int    warpSize;                   /**< Warp size in threads */
//	size_t memPitch;                   /**< Maximum pitch in bytes allowed by memory copies */
//	int    maxThreadsPerBlock;         /**< Maximum number of threads per block */
//	int    maxThreadsDim[3];           /**< Maximum size of each dimension of a block */
//	int    maxGridSize[3];             /**< Maximum size of each dimension of a grid */
//	int    clockRate;                  /**< Clock frequency in kilohertz */
//	size_t totalConstMem;              /**< Constant memory available on device in bytes */
//	int    major;                      /**< Major compute capability */
//	int    minor;                      /**< Minor compute capability */
//	size_t textureAlignment;           /**< Alignment requirement for textures */
//	size_t texturePitchAlignment;      /**< Pitch alignment requirement for texture references bound to pitched memory */
//	int    deviceOverlap;              /**< Device can concurrently copy memory and execute a kernel. Deprecated. Use instead asyncEngineCount. */
//	int    multiProcessorCount;        /**< Number of multiprocessors on device */
//	int    kernelExecTimeoutEnabled;   /**< Specified whether there is a run time limit on kernels */
//	int    integrated;                 /**< Device is integrated as opposed to discrete */
//	int    canMapHostMemory;           /**< Device can map host memory with hipHostAlloc/hipHostGetDevicePointer */
//	int    computeMode;                /**< Compute mode (See ::hipComputeMode) */
//	int    maxTexture1D;               /**< Maximum 1D texture size */
//	int    maxTexture1DMipmap;         /**< Maximum 1D mipmapped texture size */
//	int    maxTexture1DLinear;         /**< Maximum size for 1D textures bound to linear memory */
//	int    maxTexture2D[2];            /**< Maximum 2D texture dimensions */
//	int    maxTexture2DMipmap[2];      /**< Maximum 2D mipmapped texture dimensions */
//	int    maxTexture2DLinear[3];      /**< Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory */
//	int    maxTexture2DGather[2];      /**< Maximum 2D texture dimensions if texture gather operations have to be performed */
//	int    maxTexture3D[3];            /**< Maximum 3D texture dimensions */
//	int    maxTexture3DAlt[3];         /**< Maximum alternate 3D texture dimensions */
//	int    maxTextureCubemap;          /**< Maximum Cubemap texture dimensions */
//	int    maxTexture1DLayered[2];     /**< Maximum 1D layered texture dimensions */
//	int    maxTexture2DLayered[3];     /**< Maximum 2D layered texture dimensions */
//	int    maxTextureCubemapLayered[2];/**< Maximum Cubemap layered texture dimensions */
//	int    maxSurface1D;               /**< Maximum 1D surface size */
//	int    maxSurface2D[2];            /**< Maximum 2D surface dimensions */
//	int    maxSurface3D[3];            /**< Maximum 3D surface dimensions */
//	int    maxSurface1DLayered[2];     /**< Maximum 1D layered surface dimensions */
//	int    maxSurface2DLayered[3];     /**< Maximum 2D layered surface dimensions */
//	int    maxSurfaceCubemap;          /**< Maximum Cubemap surface dimensions */
//	int    maxSurfaceCubemapLayered[2];/**< Maximum Cubemap layered surface dimensions */
//	size_t surfaceAlignment;           /**< Alignment requirements for surfaces */
//	int    concurrentKernels;          /**< Device can possibly execute multiple kernels concurrently */
//	int    ECCEnabled;                 /**< Device has ECC support enabled */
//	int    pciBusID;                   /**< PCI bus ID of the device */
//	int    pciDeviceID;                /**< PCI device ID of the device */
//	int    pciDomainID;                /**< PCI domain ID of the device */
//	int    tccDriver;                  /**< 1 if device is a Tesla device using TCC driver, 0 otherwise */
//	int    asyncEngineCount;           /**< Number of asynchronous engines */
//	int    unifiedAddressing;          /**< Device shares a unified address space with the host */
//	int    memoryClockRate;            /**< Peak memory clock frequency in kilohertz */
//	int    memoryBusWidth;             /**< Global memory bus width in bits */
//	int    l2CacheSize;                /**< Size of L2 cache in bytes */
//	int    maxThreadsPerMultiProcessor;/**< Maximum resident threads per multiprocessor */
//	int    streamPrioritiesSupported;  /**< Device supports stream priorities */
//	int    globalL1CacheSupported;     /**< Device supports caching globals in L1 */
//	int    localL1CacheSupported;      /**< Device supports caching locals in L1 */
//	size_t sharedMemPerMultiprocessor; /**< Shared memory available per multiprocessor in bytes */
//	int    regsPerMultiprocessor;      /**< 32-bit registers available per multiprocessor */
//	int    managedMemory;              /**< Device supports allocating managed memory on this system */
//	int    isMultiGpuBoard;            /**< Device is on a multi-GPU board */
//	int    multiGpuBoardGroupID;       /**< Unique identifier for a group of devices on the same multi-GPU board */
//	int    hostNativeAtomicSupported;  /**< Link between the device and the host supports native atomic operations */
//	int    singleToDoublePrecisionPerfRatio; /**< Ratio of single precision performance (in floating-point operations per second) to double precision performance */
//	int    pageableMemoryAccess;       /**< Device supports coherently accessing pageable memory without calling hipHostRegister on it */
//	int    concurrentManagedAccess;    /**< Device can coherently access managed memory concurrently with the CPU */
//};
	


	cudaStatus = hipGetDeviceCount(&num);
	for (int i = 0; i < num; i++)
	{
		hipGetDeviceProperties(&prop, i);
		std::cout << "Device number: " << i << std::endl;
		std::cout << "Device name: " << prop.name << std::endl;
		std::cout << "maxThreadsPerBlock: " << prop.maxThreadsPerBlock << std::endl;
		std::cout << "multiProcessorCount : " << prop.multiProcessorCount << std::endl;
		std::cout << "blockDim(.x): " << prop.maxThreadsDim[0] <<" "
			<<prop.maxThreadsDim[1] <<" "<< prop.maxThreadsDim[2]<< std::endl;
		std::cout << "GridDim.x(.x): " << prop.maxGridSize[0] << " "
			<< prop.maxGridSize[1] << " " << prop.maxGridSize[2] << std::endl;
		std::cout << "concurrentKernels: " << prop.concurrentKernels << std::endl; // ?
		std::cout << "maxThreadsPerMultiProcessor: " << prop.maxThreadsPerMultiProcessor <<
			std::endl;	
		std::cout << "totalGlobalMem: " << prop.totalGlobalMem << std::endl; //3G
		std::cout << "major & minor: " << prop.major << " " << prop.minor << std::endl; // related to hardware
		std::cout << "WarpSize: " << prop.warpSize << std::endl;
		std::cout << "memPitch: " << prop.memPitch << std::endl;  // bytes
		std::cout << "tccDriver: " << prop.tccDriver << std::endl;
		std::cout << "singleToDoublePrecisionPerfRatio: " << prop.singleToDoublePrecisionPerfRatio << std::endl;
		std::cout << "sharedMemPerBlock: " << prop.sharedMemPerBlock << std::endl;
		

	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	

	// Free memory of MY test code partIII 3
	//freeCPU(x, y);
	hipFree(x);
	hipFree(y);

	return 0;
}